#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <stdlib.h>

__global__ void gTest(float* a, float* b, float* c){
    uint64_t i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = b[i] + a[i];
    c[i] *= c[i];
}

int main(int argc, char *argv[]){
    float *da, *ha, *fa, *host1, *host2;
    uint64_t num_of_blocks=32000, threads_per_block=atoi(argv[1]);
    uint64_t N = 32000;
    num_of_blocks = N/threads_per_block;
    host1 = (float*)(calloc(N, sizeof(float)));
    host2 = (float*)(calloc(N, sizeof(float)));
    for (int i = 0; i < N; ++i) {
        host1[i] = i;
        host2[i] = i + 3;
    }

    hipMalloc((void**)&da, N*sizeof(float));
    hipMalloc((void**)&ha, N*sizeof(float));
    hipMalloc((void**)&fa, N*sizeof(float));
    hipMemcpy(da, host1, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ha, host2, N*sizeof(float), hipMemcpyHostToDevice);
    gTest<<<dim3((uint64_t)num_of_blocks),
    dim3((uint64_t)threads_per_block)>>>(da, ha, fa);
    hipDeviceSynchronize();
    hipMemcpy(host1, fa, N*sizeof(float), hipMemcpyDeviceToHost);
    printf("%f", host1[1]);
    hipFree(da);
    hipFree(ha);
    hipFree(fa);
    free(host1);
    free(host2);
    return 0;
}