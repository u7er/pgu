#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hip/driver_types.h>

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>


#define CCHECK(ret) \
	if (_check_cuda((ret), __LINE__, __FILE__)) { \
		exit(EXIT_FAILURE); \
	}

int _check_cuda(hipError_t err, int line, const char file[]) {
	if (err != hipSuccess) {
		fprintf(stderr, "%s, %d in %s\n", hipGetErrorString(err), line, file);
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}

__global__ void Init(float *a) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	a[i] = i;
}

int main(int argc, char **argv) {
	float *da, *ha;
	int nob = atoi(argv[1]), tpb = 1024;
	int N = nob * tpb;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsedTime = 0;

	ha = (float *) calloc(N, sizeof(float));
	CCHECK(hipMalloc((void ** ) &da, N * sizeof(float)));

	hipEventRecord(start, 0);
	Init<<<dim3(nob), dim3(tpb)>>>(da);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	//CCHECK(cudaDeviceSynchronize());

	CCHECK(hipGetLastError());
	CCHECK(hipMemcpy(ha, da, N * sizeof(float), hipMemcpyDeviceToHost));
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Elapsed time %gms\n", elapsedTime);

	for (int i = 0; i < N; ++i) {
		// printf("%d%c", ha[i], (i % 30 == 0 || i == (N - 1)? '\n' : ' '));
	}
	printf("End\n");
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(ha);
	CCHECK(hipFree(da));
	return 0;
}
